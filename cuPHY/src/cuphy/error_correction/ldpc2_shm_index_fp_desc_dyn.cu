#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2023, NVIDIA CORPORATION & AFFILIATES.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 */


//#define CUPHY_DEBUG 1

#include "ldpc2_desc.cuh"
#include "ldpc2_c2v.cuh"
#include "ldpc2_app_address_fp_desc.cuh"
#include "ldpc2_app_address_fp_dp_desc.cuh"
#include "ldpc2_min_sum_update_half_0.cuh"
#include "ldpc2_shm_index_fp_desc_dyn.hpp"
#include "ldpc2_schedule_dynamic_desc.cuh"
#include "ldpc2_c2v_cache_shared.cuh"

using namespace ldpc2;

#define USE_APP_ADDR_FP_DP 1

namespace
{
    // Single set of values for all kernels in this module, for now...
    const int MAX_THREADS_PER_CTA = 384;
    const int MIN_CTA_PER_SM      = 1;
    const int NUM_STORAGE_WORDS   = 2;
    const int MIN_PARITY_ROWS     = 4;

    // Base graph-dependent maximum number of parity nodes. Limited by
    // the amount of shared memory...
    template <int BG> struct shm_max_parity_nodes;
    template <>
    struct shm_max_parity_nodes<1>
    {
        static const int value = 20;
    };
    template <>
    struct shm_max_parity_nodes<2>
    {
        static const int value = 20;
    };

    //------------------------------------------------------------------
    // Sign manager policies (for compressed C2V row processors)
    typedef ldpc2::sign_store_policy_split_dst<__half, ldpc2::split_sign_update_fp,      false> sign_dst_fp_t;
    typedef ldpc2::sign_store_policy_split_dst<__half, ldpc2::split_sign_update_bit_ops, false> sign_dst_bit_t;
    typedef ldpc2::sign_store_policy_split_src<__half, ldpc2::split_sign_update_fp,      false> sign_src_fp_t;
    typedef ldpc2::sign_store_policy_split_src<__half, ldpc2::split_sign_update_bit_ops, false> sign_src_bit_t;
    
    // Sign updates with FP unit A couple of microseconds faster on V100...
    typedef sign_dst_fp_t                                                                       sign_mgr_t;

    // APP address calculation
    // Using floating point instruction APP address calculation
    // sequence
#if USE_APP_ADDR_FP_DP
    template <int BG> using app_loc_t = app_loc_address_fp_dp_desc<__half, BG>;
#else
    template <int BG> using app_loc_t = app_loc_address_fp_desc<__half, BG>;
#endif

    // C2V per-row storage. Larger storage allows faster row
    // processing, but increases register pressure (and may incur
    // register spills).
    typedef ldpc2::C2V_storage_t<__half, NUM_STORAGE_WORDS>                                     c2v_storage_t;

    //------------------------------------------------------------------
    // Alias template for compressed C2V row processors, with a template
    // parameter for the C2V row storage. The sign processor and min
    // sum updater have been chosen to be the "fastest" for some
    // architecture and lifting size combinations.
    typedef ldpc2::cC2V_row_proc<__half,
                                 ldpc2::cC2V_row_context<__half,
                                                         sign_dst_fp_t,
                                                         ldpc2::min_sum_update_half_0,
                                                         c2v_storage_t>
                                > cC2V_row_proc;
    //------------------------------------------------------------------
    // cC2V_all_row_map_t
    // The C2V_row_proc template requires a row map template with template
    // arguments BG (int), CHECK_IDX (int), TStorage (per-row storage
    // structure.
    template <int   BG,
              int   CHECK_IDX,
              class TC2VStorage> using cC2V_all_row_map_t = simple_row_map<BG,
                                                                           CHECK_IDX,
                                                                           TC2VStorage,
                                                                           cC2V_row_proc>;
    //------------------------------------------------------------------
    // Kernel configuration structure, with typedefs for kernel execution
    // BG_: base graph (1 or 2)
    // TKernelParams: Class/struct used for kernel parameters
    // TLLR: Source LLR data type (__half or float)
    // TLoader: LLR loader template struct (e.g. llr_loader_batch)
    template <int                                                                BG_,
              class                                                              TKernelParams,
              typename                                                           TLLR,
              template <typename, int, template<typename, typename> class> class TLoader>
    struct ldpc2_shm_index_fp_desc_dyn_kernel_config
    {
        static constexpr int BG              = BG_;
        static constexpr int MAX_PARITY_ROWS = shm_max_parity_nodes<BG>::value;

        typedef TKernelParams                   kernel_params_t;
        
        typedef C2V_row_proc<__half,
                             BG,
                             cC2V_all_row_map_t,
                             app_loader,
                             app_writer>                                  C2V_t;
        //- - - - - - - - - - - - - - - - - - - - - - - - - - - - - - -
        // C2V message cache (TODO: use shared memory here)
        typedef ldpc2::c2v_cache_shared<BG,
                                        C2V_t,
                                        c2v_storage_t,
                                        kernel_params_t>                        c2v_cache_t;
        //- - - - - - - - - - - - - - - - - - - - - - - - - - - - - - -
        // LLR loader, used to load LLR data from global to shared memory
        typedef TLoader<TLLR, 4, llr_op_clamp>                                  llr_loader_t;
        // Data type in APP shared memory buffer (__half or __half2)
        typedef typename llr_loader_t::app_buf_t                                app_buf_t;
        //- - - - - - - - - - - - - - - - - - - - - - - - - - - - - - -
        // "Dynamic" schedule, with the number of parity rows not known until runtime.
        typedef ldpc2::ldpc_schedule_dynamic_desc<BG,
                                                  app_loc_t<BG>,
                                                  c2v_cache_t,
                                                  kernel_params_t,
                                                  typename app_loc_t<BG>::bg_desc_t,
                                                  MIN_PARITY_ROWS,
                                                  MAX_PARITY_ROWS> sched_t;
    };

    //------------------------------------------------------------------
    // get_shmem_required()
    // For the shared memory kernel, this is the sum of the APP storage
    // and the C2V data storage.
    int get_shmem_required(int BG,
                           int num_parity_nodes,
                           int Z)
    {
        const uint32_t NUM_VAR_NODES = ((1 == BG) ? ldpc2::max_info_nodes<1>::value : ldpc2::max_info_nodes<2>::value) +
                                       num_parity_nodes;
        const int APP_SIZE = static_cast<int>(shmem_llr_buffer_size(NUM_VAR_NODES,    // num shared memory nodes
                                                                    Z,                // lifting size
                                                                    sizeof(__half))); // element size
        const int C2V_SIZE = (num_parity_nodes * Z * sizeof(c2v_storage_t));
        // We need to pad the APP portion of shared memory to make sure
        // that C2V storage is aligned with the C2V type.
        const int SHMEM_SIZE = round_up_to_next(APP_SIZE, static_cast<int>(alignof(c2v_storage_t))) +
                                                C2V_SIZE;
        return SHMEM_SIZE;
    }
} // namespace


////////////////////////////////////////////////////////////////////////
// ldpc2_BG1_shm_index_fp_desc_dyn()
// Base graph 1 kernel, "legacy" tensor interface
extern "C"
__global__ __launch_bounds__(MAX_THREADS_PER_CTA, MIN_CTA_PER_SM)
void ldpc2_BG1_shm_index_fp_desc_dyn(LDPC_kernel_params params, app_loc_t<1>::bg_desc_t bgdesc)
{
    // Shared memory is allocated dynamically
    extern __shared__ char smem[];

    //------------------------------------------------------------------
    // Kernel configuration template
    typedef ldpc2_shm_index_fp_desc_dyn_kernel_config<1,
                                                      ldpc2::LDPC_kernel_params,
                                                      __half,
                                                      ldpc2::llr_loader_variable_batch> kernel_config_t;

    //------------------------------------------------------------------
    // Load LLR data from global to shared memory
    kernel_config_t::llr_loader_t::load_sync(smem, params, blockIdx.x);

    //------------------------------------------------------------------
    // Perform iterations
    kernel_config_t::sched_t sched(smem,
                                   params,
                                   bgdesc,
                                   static_cast<int>(__cvta_generic_to_shared(smem)),
                                   threadIdx.x);
    for(int iter = 0; iter < params.max_iterations; ++iter)
    {
        sched.do_iteration();
    }

    //------------------------------------------------------------------
    // Write hard output based on APP values
    //ldpc_dec_output_variable(params, reinterpret_cast<const kernel_config_t::app_buf_t*>(smem));
    ldpc_dec_output_variable_loop(params, reinterpret_cast<const kernel_config_t::app_buf_t*>(smem));
}

////////////////////////////////////////////////////////////////////////
// ldpc2_BG2_shm_index_fp_desc_dyn()
// Base graph 2 kernel, "legacy" tensor interface
extern "C"
__global__ __launch_bounds__(MAX_THREADS_PER_CTA, MIN_CTA_PER_SM)
void ldpc2_BG2_shm_index_fp_desc_dyn(LDPC_kernel_params params, app_loc_t<2>::bg_desc_t bgdesc)
{
    // Shared memory is allocated dynamically
    extern __shared__ char smem[];
    
    //------------------------------------------------------------------
    // Kernel configuration template
    typedef ldpc2_shm_index_fp_desc_dyn_kernel_config<2, // BG
                                                      ldpc2::LDPC_kernel_params,
                                                      __half,
                                                      ldpc2::llr_loader_variable_batch> kernel_config_t;

    //------------------------------------------------------------------
    // Load LLR data from global to shared memory
    kernel_config_t::llr_loader_t::load_sync(smem, params, blockIdx.x);

    //------------------------------------------------------------------
    // Perform iterations
    kernel_config_t::sched_t sched(smem,
                                   params,
                                   bgdesc,
                                   static_cast<int>(__cvta_generic_to_shared(smem)),
                                   threadIdx.x);
    for(int iter = 0; iter < params.max_iterations; ++iter)
    {
        sched.do_iteration();
    }

    //------------------------------------------------------------------
    // Write hard output based on APP values
    ldpc_dec_output_variable(params, reinterpret_cast<const kernel_config_t::app_buf_t*>(smem));
    // No loop needed for BG2 with Z>= 32
    //ldpc_dec_output_variable_loop(params, reinterpret_cast<const kernel_config_t::app_buf_t*>(smem));
}

////////////////////////////////////////////////////////////////////////
// ldpc2_BG1_shm_index_fp_desc_dyn_tb()
// Base graph 1 kernel, transport block interface
extern "C"
__global__ __launch_bounds__(MAX_THREADS_PER_CTA, MIN_CTA_PER_SM)
void ldpc2_BG1_shm_index_fp_desc_dyn_tb(cuphyLDPCDecodeDesc_t decodeDesc, app_loc_t<1>::bg_desc_t bgdesc)
{
    // Shared memory is allocated dynamically
    extern __shared__ char smem[];

    //------------------------------------------------------------------
    // Kernel configuration template
    typedef ldpc2_shm_index_fp_desc_dyn_kernel_config<1,
                                                      cuphyLDPCDecodeConfigDesc_t,
                                                      __half,
                                                      ldpc2::llr_loader_variable_batch> kernel_config_t;

    //------------------------------------------------------------------
    // Load LLR data from global to shared memory
    kernel_config_t::llr_loader_t::load_sync(smem, decodeDesc, blockIdx.x);

    //------------------------------------------------------------------
    // Perform iterations
    kernel_config_t::sched_t sched(smem,
                                   decodeDesc.config,
                                   bgdesc,
                                   static_cast<int>(__cvta_generic_to_shared(smem)),
                                   threadIdx.x);
    for(int iter = 0; iter < decodeDesc.config.max_iterations; ++iter)
    {
        sched.do_iteration();
    }

    //------------------------------------------------------------------
    // Write hard output based on APP values
    //ldpc_dec_output_variable(decodeDesc, reinterpret_cast<const kernel_config_t::app_buf_t*>(smem));
    ldpc_dec_output_variable_loop(decodeDesc, reinterpret_cast<const kernel_config_t::app_buf_t*>(smem));
}

////////////////////////////////////////////////////////////////////////
// ldpc2_BG2_shm_index_fp_desc_dyn_tb()
// Base graph 2 kernel, transport block interface
extern "C"
__global__ __launch_bounds__(MAX_THREADS_PER_CTA, MIN_CTA_PER_SM)
void ldpc2_BG2_shm_index_fp_desc_dyn_tb(cuphyLDPCDecodeDesc_t decodeDesc, app_loc_t<2>::bg_desc_t bgdesc)
{
    // Shared memory is allocated dynamically
    extern __shared__ char smem[];
    
    //------------------------------------------------------------------
    // Kernel configuration template
    typedef ldpc2_shm_index_fp_desc_dyn_kernel_config<2, // BG
                                                      cuphyLDPCDecodeConfigDesc_t,
                                                      __half,
                                                      ldpc2::llr_loader_variable_batch> kernel_config_t;

    //------------------------------------------------------------------
    // Load LLR data from global to shared memory
    kernel_config_t::llr_loader_t::load_sync(smem, decodeDesc, blockIdx.x);

    //------------------------------------------------------------------
    // Perform iterations
    kernel_config_t::sched_t sched(smem,
                                   decodeDesc.config,
                                   bgdesc,
                                   static_cast<int>(__cvta_generic_to_shared(smem)),
                                   threadIdx.x);
    for(int iter = 0; iter < decodeDesc.config.max_iterations; ++iter)
    {
        sched.do_iteration();
    }

    //------------------------------------------------------------------
    // Write hard output based on APP values
    ldpc_dec_output_variable(decodeDesc, reinterpret_cast<const kernel_config_t::app_buf_t*>(smem));
    // No loop needed for BG2 with Z>= 32
    //ldpc_dec_output_variable_loop(decodeDesc, reinterpret_cast<const kernel_config_t::app_buf_t*>(smem));
}

namespace ldpc2
{

////////////////////////////////////////////////////////////////////////
// shm_index_fp_desc_dyn::decode()
cuphyStatus_t shm_index_fp_desc_dyn::decode(ldpc::decoder&                     dec,
                                            LDPC_output_t&                     tDst,
                                            const_tensor_pair&                 tLLR,
                                            const cuphyLDPCDecodeConfigDesc_t& config,
                                            hipStream_t                       strm)
{
    DEBUG_PRINTF("ldpc2::shm_index_fp_desc_dyn::decode()\n");
    //------------------------------------------------------------------
    cuphyDataType_t llrType = tLLR.first.get().type();
    const int       NUM_CW  = tLLR.first.get().layout().dimensions[1];
    //------------------------------------------------------------------
    dim3 grdDim(NUM_CW);
    dim3 blkDim(config.Z);

    //------------------------------------------------------------------
    // Initialize the kernel params struct
    LDPC_kernel_params params(config, tLLR, tDst);

    cuphyStatus_t s = CUPHY_STATUS_NOT_SUPPORTED;
    
    //------------------------------------------------------------------
    // Determine the dynamic amount of shared memory
    const uint32_t SHMEM_SIZE = get_shmem_required(config.BG,
                                                   config.num_parity_nodes,
                                                   config.Z);
    if(llrType == CUPHY_R_16F)
    {
        switch(config.BG)
        {
        case 1:
            {
                //-  -  -  -  -  -  -  -  -  -  -  -  -  -  -  -  -  -
                // Retrieve the base graph descriptor
                const app_loc_t<1>::bg_desc_t* bgdesc = app_loc_t<1>::get_bg_desc(params.Z);
                if(!bgdesc) break;
                
                DEBUG_PRINT_FUNC_MAX_BLOCKS(ldpc2_BG1_shm_index_fp_desc_dyn, blkDim, SHMEM_SIZE);
                
                //-  -  -  -  -  -  -  -  -  -  -  -  -  -  -  -  -  -
                // Launch the kernel
                ldpc2_BG1_shm_index_fp_desc_dyn<<<grdDim, blkDim, SHMEM_SIZE, strm>>>(params, *bgdesc);
                s = CUPHY_STATUS_SUCCESS;

            }
            break;
        case 2:
            {
                //-  -  -  -  -  -  -  -  -  -  -  -  -  -  -  -  -  -
                // Retrieve the base graph descriptor
                const app_loc_t<2>::bg_desc_t* bgdesc = app_loc_t<2>::get_bg_desc(params.Z);
                if(!bgdesc) break;

                DEBUG_PRINT_FUNC_MAX_BLOCKS(ldpc2_BG2_shm_index_fp_desc_dyn, blkDim, SHMEM_SIZE);
                
                //-  -  -  -  -  -  -  -  -  -  -  -  -  -  -  -  -  -
                // Launch the kernel
                ldpc2_BG2_shm_index_fp_desc_dyn<<<grdDim, blkDim, SHMEM_SIZE, strm>>>(params, *bgdesc);
                s = CUPHY_STATUS_SUCCESS;
            }
            break;
        default:
            break;
        }
    }
    
    if(CUPHY_STATUS_SUCCESS != s)
    {
        return s;
    }

#if CUPHY_DEBUG
    hipDeviceSynchronize();
#endif
    hipError_t e = hipGetLastError();
    DEBUG_PRINTF("CUDA STATUS (%s:%i): %s\n", __FILE__, __LINE__, hipGetErrorString(e));
    return (e == hipSuccess) ? CUPHY_STATUS_SUCCESS : CUPHY_STATUS_INTERNAL_ERROR;
}

////////////////////////////////////////////////////////////////////////
// shm_index_fp_desc_dyn::decode_tb()
cuphyStatus_t shm_index_fp_desc_dyn::decode_tb(ldpc::decoder&               dec,
                                               const cuphyLDPCDecodeDesc_t& decodeDesc,
                                               hipStream_t                 strm)
{
    DEBUG_PRINTF("ldpc2::shm_index_fp_desc_dyn::decode_tb()\n");    
    cuphyStatus_t s = CUPHY_STATUS_NOT_SUPPORTED;

    //------------------------------------------------------------------
    dim3 grdDim(ldpc::decoder::get_total_num_codewords(decodeDesc));
    dim3 blkDim(decodeDesc.config.Z);

    //------------------------------------------------------------------
    // Determine the dynamic amount of shared memory
    const uint32_t SHMEM_SIZE = get_shmem_required(decodeDesc.config.BG,
                                                   decodeDesc.config.num_parity_nodes,
                                                   decodeDesc.config.Z);
    if(decodeDesc.config.llr_type == CUPHY_R_16F)
    {
        switch(decodeDesc.config.BG)
        {
        case 1:
            {
                //------------------------------------------------------------------
                // Retrieve the base graph descriptor
                const app_loc_t<1>::bg_desc_t* bgdesc = app_loc_t<1>::get_bg_desc(decodeDesc.config.Z);
                if(!bgdesc) break;
                
                DEBUG_PRINT_FUNC_MAX_BLOCKS(ldpc2_BG1_shm_index_fp_desc_dyn_tb, blkDim, SHMEM_SIZE);

                //------------------------------------------------------------------
                // Launch the kernel
                ldpc2_BG1_shm_index_fp_desc_dyn_tb<<<grdDim, blkDim, SHMEM_SIZE, strm>>>(decodeDesc, *bgdesc);
                s = CUPHY_STATUS_SUCCESS;
            }
            break;
        case 2:
            {
                //------------------------------------------------------------------
                // Retrieve the base graph descriptor
                const app_loc_t<2>::bg_desc_t* bgdesc = app_loc_t<2>::get_bg_desc(decodeDesc.config.Z);
                if(!bgdesc) break;
                
                DEBUG_PRINT_FUNC_MAX_BLOCKS(ldpc2_BG2_shm_index_fp_desc_dyn_tb, blkDim, SHMEM_SIZE);
                
                //------------------------------------------------------------------
                // Launch the kernel
                ldpc2_BG2_shm_index_fp_desc_dyn_tb<<<grdDim, blkDim, SHMEM_SIZE, strm>>>(decodeDesc, *bgdesc);
                s = CUPHY_STATUS_SUCCESS;
            }
            break;
        default:
            break;
        }
    }
    if(CUPHY_STATUS_SUCCESS != s)
    {
        return s;
    }

#if CUPHY_DEBUG
    hipDeviceSynchronize();
#endif
    hipError_t e = hipGetLastError();
    DEBUG_PRINTF("CUDA STATUS (%s:%i): %s\n", __FILE__, __LINE__, hipGetErrorString(e));
    return (e == hipSuccess) ? CUPHY_STATUS_SUCCESS : CUPHY_STATUS_INTERNAL_ERROR;
}

////////////////////////////////////////////////////////////////////////
// shm_index_fp_desc_dyn::get_workspace_size()
std::pair<bool, size_t> shm_index_fp_desc_dyn::get_workspace_size(const ldpc::decoder&               dec,
                                                                  const cuphyLDPCDecodeConfigDesc_t& config,
                                                                  int                                num_cw)
{
    return std::pair<bool, size_t>(true, 0);
}

////////////////////////////////////////////////////////////////////////
// shm_index_fp_x2_desc_dyn::can_decode_config()
bool shm_index_fp_desc_dyn::can_decode_config(const ldpc::decoder&               dec,
                                              const cuphyLDPCDecodeConfigDesc_t& cfg)
{
    // Compare shared memory requirements to device maximum
    return (get_shmem_required(cfg.BG, cfg.num_parity_nodes, cfg.Z) <= dec.max_shmem_per_block_optin());
}

////////////////////////////////////////////////////////////////////////
// shm_index_fp_desc_dyn::shm_index_fp_desc_dyn()
shm_index_fp_desc_dyn::shm_index_fp_desc_dyn(ldpc::decoder& dec)
{
    //------------------------------------------------------------------
    // Determine the maximum shared memory size that a launch might
    // require, and set the kernel attribute accordingly.
    const int MAX_BG1_SHMEM_SIZE = std::min(get_shmem_required(1,
                                                               shm_max_parity_nodes<1>::value,
                                                               CUPHY_LDPC_MAX_LIFTING_SIZE),
                                            dec.max_shmem_per_block_optin());
    const int MAX_BG2_SHMEM_SIZE = std::min(get_shmem_required(2,
                                                               shm_max_parity_nodes<2>::value,
                                                               CUPHY_LDPC_MAX_LIFTING_SIZE),
                                            dec.max_shmem_per_block_optin());
    //------------------------------------------------------------------
    // For each kernel, set the maximum dynamic shared memory size
    typedef std::pair<const void*, int> func_attr_t;
    std::array<func_attr_t, 4> func_attrs =
    {
        func_attr_t((const void*)ldpc2_BG1_shm_index_fp_desc_dyn,    MAX_BG1_SHMEM_SIZE),
        func_attr_t((const void*)ldpc2_BG2_shm_index_fp_desc_dyn,    MAX_BG2_SHMEM_SIZE),
        func_attr_t((const void*)ldpc2_BG1_shm_index_fp_desc_dyn_tb, MAX_BG1_SHMEM_SIZE),
        func_attr_t((const void*)ldpc2_BG2_shm_index_fp_desc_dyn_tb, MAX_BG2_SHMEM_SIZE)
    };
    for(func_attr_t f_a : func_attrs)
    {
        hipError_t e = hipFuncSetAttribute(reinterpret_cast<const void*>(f_a.first),
                                             hipFuncAttributeMaxDynamicSharedMemorySize,
                                             f_a.second);
        if(hipSuccess != e)
        {
            throw cuphy_i::cuda_exception(e);
        }
    }
    //------------------------------------------------------------------
    DEBUG_PRINT_FUNC_ATTRIBUTES(ldpc2_BG1_shm_index_fp_desc_dyn);
    DEBUG_PRINT_FUNC_ATTRIBUTES(ldpc2_BG2_shm_index_fp_desc_dyn);
    DEBUG_PRINT_FUNC_ATTRIBUTES(ldpc2_BG1_shm_index_fp_desc_dyn_tb);
    DEBUG_PRINT_FUNC_ATTRIBUTES(ldpc2_BG2_shm_index_fp_desc_dyn_tb);
}

////////////////////////////////////////////////////////////////////////
// shm_index_fp_desc_dyn::get_launch_config()
cuphyStatus_t shm_index_fp_desc_dyn::get_launch_config(const ldpc::decoder&           dec,
                                                       cuphyLDPCDecodeLaunchConfig_t& launchConfig)
{
    const int Z                = launchConfig.decode_desc.config.Z;
    const int BG               = launchConfig.decode_desc.config.BG;
    const int NUM_PARITY_NODES = launchConfig.decode_desc.config.num_parity_nodes;
    const int MAX_PARITY_NODES = (1 == BG)                  ?
                                 max_parity_nodes<1>::value :
                                 max_parity_nodes<2>::value;
    const int NUM_VAR_NODES    = ldpc::decoder::get_num_variable_nodes(BG,
                                                                       NUM_PARITY_NODES);
    //------------------------------------------------------------------
    // Validate input arguments
    if((Z < 2)                              ||
       (Z > CUPHY_LDPC_MAX_LIFTING_SIZE)    ||
       (NUM_PARITY_NODES < 4)               ||
       (NUM_PARITY_NODES > MAX_PARITY_NODES))
    {
        return CUPHY_STATUS_UNSUPPORTED_CONFIG;
    }
    
    const uint32_t SHMEM_SIZE = get_shmem_required(launchConfig.decode_desc.config.BG,
                                                   launchConfig.decode_desc.config.num_parity_nodes,
                                                   launchConfig.decode_desc.config.Z);

    //------------------------------------------------------------------
    // Set up launch geometry and the kernel function (driver)
    #if CUDART_VERSION >= 11000
    launchConfig.kernel_node_params_driver.blockDimX = Z;
    launchConfig.kernel_node_params_driver.blockDimY = 1;
    launchConfig.kernel_node_params_driver.blockDimZ = 1;

    launchConfig.kernel_node_params_driver.gridDimX = ldpc::decoder::get_total_num_codewords(launchConfig.decode_desc);
    launchConfig.kernel_node_params_driver.gridDimY = 1;
    launchConfig.kernel_node_params_driver.gridDimZ = 1;

    launchConfig.kernel_node_params_driver.extra          = nullptr;
    launchConfig.kernel_node_params_driver.kernelParams   = launchConfig.kernel_args;
    launchConfig.kernel_node_params_driver.sharedMemBytes = SHMEM_SIZE;

    hipFunction_t deviceFunction;
    hipError_t    e = (BG == 1) ?  hipGetFuncBySymbol(&deviceFunction, (void*)ldpc2_BG1_shm_index_fp_desc_dyn_tb): 
                                    hipGetFuncBySymbol(&deviceFunction, (void*)ldpc2_BG2_shm_index_fp_desc_dyn_tb);
    if (e != hipSuccess) 
    {
        return CUPHY_STATUS_INTERNAL_ERROR;
    }
    launchConfig.kernel_node_params_driver.func = static_cast<hipFunction_t>(deviceFunction);
    #endif
    //------------------------------------------------------------------
    // Set kernel arguments:
    // arg 0: decode descriptor
    launchConfig.kernel_args[0] = &launchConfig.decode_desc;
    // arg 1: base graph descriptor
    if(1 == BG)
    {
        const app_loc_t<1>::bg_desc_t* bgdesc = app_loc_t<1>::get_bg_desc(Z);
        launchConfig.kernel_args[1] = const_cast<void*>(reinterpret_cast<const void*>(bgdesc));
    }
    else
    {
        const app_loc_t<2>::bg_desc_t* bgdesc = app_loc_t<2>::get_bg_desc(Z);
        launchConfig.kernel_args[1] = const_cast<void*>(reinterpret_cast<const void*>(bgdesc));
    }
    return CUPHY_STATUS_SUCCESS;
}

} // namespace ldpc2
