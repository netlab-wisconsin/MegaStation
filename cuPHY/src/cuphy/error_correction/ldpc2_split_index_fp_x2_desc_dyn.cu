#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2023, NVIDIA CORPORATION & AFFILIATES.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 */


//#define CUPHY_DEBUG 1

#include "ldpc2_c2v_x2.cuh"
#include "ldpc2_app_address_fp_dp_desc.cuh"
#include "ldpc2_app_address_dp_desc.cuh"
#include "ldpc2_schedule_dynamic_desc.cuh"
#include "nrLDPC_templates.cuh"
#include "ldpc2_desc.cuh"
#include "ldpc2_split_index_fp_x2_desc_dyn.hpp"
#include "ldpc2_c2v_cache_split.cuh"

#define LDPC_DECODE_USE_TB_SCAN 1

using namespace ldpc2;

namespace
{
    // Single set of values for all kernels in this module, for now...
    const int MAX_THREADS_PER_CTA = 384;
    const int MIN_CTA_PER_SM      = 1;

    //------------------------------------------------------------------
    // Storing compressed check to variable (cC2V) data in registers may
    // not be possible for all code rates. Furthermore, squeezing a
    // larger number of parity node data into registers may actually
    // decrease performance at high code rates.
    template <int BG> struct num_reg_parity;
    template <> struct num_reg_parity<1> { static constexpr int value = 33; };
    template <> struct num_reg_parity<2> { static constexpr int value = 42; };
    template <int BG> struct max_num_parity;
    template <> struct max_num_parity<1> { static constexpr int value = 46; };
    template <> struct max_num_parity<2> { static constexpr int value = 42; };
    
    //------------------------------------------------------------------
    // Sign manager for compressed C2V row processor
    typedef sign_mgr_pair_src<false> sign_mgr_t;

    //------------------------------------------------------------------
    // APP address calculation
    // Using floating point with dot product instruction sequence for
    // this decoder algorithm. Note that the base graph descriptor
    // argument to the kernel needs to be the "adjusted" descriptor
    // structure.
    template <int BG> using app_loc_t = app_loc_address_fp_dp_desc<__half2, BG>;
    // slightly slower on sm86
    //template <int BG> using app_loc_t = app_loc_address_dp_desc<__half2, BG>;

    //------------------------------------------------------------------
    // Template alias for a half2 row context, templated ONLY on the
    // underlying storage type. (For this decoder, we will use different
    // row contexts, and thus slightly different row processors,  for
    // the "high degree" core rows.)
    template <class TStorage> using row_context_t = cC2V_row_context<__half2,
                                                                     sign_mgr_t,
                                                                     unused,
                                                                     TStorage>;
    //------------------------------------------------------------------
    // Template alias for a half2 compressed C2V row processors,
    // templated ONLY on the row context used. This will be used by the
    // row mappers, which will instantiate a cC2V_row_proc_t template
    // instance for the different row context storage types.
    template <class TRowContext> using cC2V_row_proc_t = cC2V_row_proc<__half2,
                                                                       TRowContext>;

    //------------------------------------------------------------------
    // Kernel configuration structure, with typedefs for kernel execution
    //
    // Better perf at very high code rates when the MAX_PARITY_NODES
    // is smaller, but for now we'll prefer to get 2X codewords for
    // as many parity nodes as possible. (Try 32 vs. 28 to see the perf
    // difference.)
    // TODO: small, med, large parity count kernels?
    template <int   BG_,                 // base graph (1 or 2)
              class TKernelParams>       // struct with kernel params
    struct ldpc2_split_index_fp_x2_desc_dyn_kernel_config
    {
        static constexpr int BG                  = BG_;
        static constexpr int MIN_PARITY_ROWS     = 4;
        static constexpr int NUM_REG_PARITY_ROWS = num_reg_parity<BG>::value;
        static constexpr int MAX_PARITY_ROWS     = max_num_parity<BG>::value;
        
        typedef TKernelParams                           kernel_params_t;

        //- - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - 
        // cC2V_storage_row_map_t
        // The C2V_row_proc template requires a row map template with template
        // arguments BG (int), CHECK_IDX (int), TStorage (per-row storage
        // structure.
        template <int   BG,
                  int   CHECK_IDX,
                  class TC2VStorage> using cC2V_row_map_t = context_storage_row_map<BG,
                                                                                    CHECK_IDX,
                                                                                    TC2VStorage,
                                                                                    __half2,
                                                                                    row_context_t,
                                                                                    cC2V_row_proc_t>;
        //- - - - - - - - - - - - - - - - - - - - - - - - - - - - - - -
        // C2V row dispatch type: uses the row map to determine which
        // C2V processor to call for each row.
        typedef C2V_row_proc<__half2,
                             BG,
                             cC2V_row_map_t,
                             app_loader,
                             app_writer> C2V_t;
        //- - - - - - - - - - - - - - - - - - - - - - - - - - - - - - -
        // C2V message cache (split between register and shared memory
        // here). Two storage types are provided: one for the "core"
        // parity rows, and one for the "non-core" rows. For BG1, the
        // core rows are high-degree (19) and the rest are low-degree
        // (10 or less). For BG2, all rows are low-degree (10 or less).
        typedef ldpc2::c2v_cache_split<BG,
                                       NUM_REG_PARITY_ROWS,
                                       C2V_t,
                                       typename core_storage_x2<BG>::type, // core cC2V storage
                                       cC2V_storage_x2_low_degree,         // non-core cC2V storage
                                       kernel_params_t> c2v_cache_t;
        //- - - - - - - - - - - - - - - - - - - - - - - - - - - - - - -
        // LLR loader, used to load LLR data from global to shared memory
        typedef ldpc2::llr_loader_variable_batch<__half2, 4, llr_op_clamp> llr_loader_t;
        // Data type in APP shared memory buffer (__half or __half2)
        typedef llr_loader_t::app_buf_t                                    app_buf_t;
        //- - - - - - - - - - - - - - - - - - - - - - - - - - - - - - -
        // "Dynamic" schedule, with the number of parity rows not known until runtime.
        typedef ldpc2::ldpc_schedule_dynamic_desc<BG,
                                                  app_loc_t<BG>,
                                                  c2v_cache_t,
                                                  kernel_params_t,
                                                  typename app_loc_t<BG_>::bg_desc_t,
                                                  MIN_PARITY_ROWS,
                                                  MAX_PARITY_ROWS> sched_t;
    };
    //------------------------------------------------------------------
    // get_app_c2v_shmem()
    // Returns the number of bytes required for APP and C2V memory for
    // this kernel.
    template <int BG>
    CUDA_BOTH
    int get_app_c2v_shmem(int num_parity_nodes, int Z)
    {
        const     int32_t NUM_VAR_NODES = ldpc2::max_info_nodes<BG>::value + num_parity_nodes;
        constexpr int32_t NUM_REG_NODES = num_reg_parity<BG>::value;
        const     int32_t APP_SIZE      = static_cast<int32_t>(shmem_llr_buffer_size(NUM_VAR_NODES,     // num shared memory nodes
                                                                                     Z,                 // lifting size
                                                                                     sizeof(__half2))); // element size
        // The first 'NUM_REG_NODES' of C2V data will reside in registers.
        // The remainder will be in shared memory.
        const int32_t C2V_SIZE = (num_parity_nodes > NUM_REG_NODES)                                          ?
                                 (num_parity_nodes - NUM_REG_NODES) * Z * sizeof(cC2V_storage_x2_low_degree) :
                                 0;
        // We need to pad the APP portion of shared memory to make sure
        // that C2V storage is aligned with the C2V type.
        int shmem_size = round_up_to_next(APP_SIZE, static_cast<int>(alignof(cC2V_storage_x2_low_degree))) +
                                          C2V_SIZE;
        return shmem_size;
    }
    //------------------------------------------------------------------
    // get_shmem_required()
    // Calculates the sum of the APP and C2V data storage.
    int get_shmem_required(int BG,
                           int num_parity_nodes,
                           int Z)
    {
        int shmem_size = (1 == BG) ? get_app_c2v_shmem<1>(num_parity_nodes, Z)
                                   : get_app_c2v_shmem<2>(num_parity_nodes, Z);
#if LDPC_DECODE_USE_TB_SCAN
        // When using a scan algorithm to determine the codeword for a CTA,
        // extra shared memory for the token is required.
        shmem_size = round_up_to_next(shmem_size, static_cast<int>(alignof(tb_token))) +
                     sizeof(tb_token);
#endif
        return shmem_size;
    }
#if LDPC_DECODE_USE_TB_SCAN
    //------------------------------------------------------------------
    // get_token_addr()
    // Returns the address of the tb_token value used to store information
    // about the specific codeword being processed by a CTA when the
    // transport block interface is used. The token is assumed to reside
    // immediately after the APP and C2V memory.
    template <int BG>
    __device__
    tb_token* get_token_addr(int num_parity_nodes, int Z, char* smem)
    {
        return reinterpret_cast<tb_token*>(smem + get_app_c2v_shmem<BG>(num_parity_nodes, Z));
    }
    template <int BG>
    __device__
    tb_token* get_token_addr(const cuphyLDPCDecodeDesc_t& decodeDesc,
                             char* smem)
    {
        return get_token_addr<BG>(decodeDesc.config.num_parity_nodes,
                                  decodeDesc.config.Z,
                                  smem);
    }
#endif // if LDPC_DECODE_USE_TB_SCAN
} // namespace

////////////////////////////////////////////////////////////////////////
// ldpc2_BG1_split_index_fp_x2_desc_dyn()
// Kernel for base graph 1 (legacy tensor interface)
extern "C"
__global__ __launch_bounds__(MAX_THREADS_PER_CTA, MIN_CTA_PER_SM)
void ldpc2_BG1_split_index_fp_x2_desc_dyn(LDPC_kernel_params params, app_loc_t<1>::bg_desc_t bgdesc)
{
    // Shared memory is allocated dynamically
    extern __shared__ char smem[];

    //------------------------------------------------------------------
    // Kernel configuration template
    typedef ldpc2_split_index_fp_x2_desc_dyn_kernel_config<1,                                          // BG
                                                           ldpc2::LDPC_kernel_params> kernel_config_t; // params struct

    //------------------------------------------------------------------
    // Load LLR data from global to shared memory
    kernel_config_t::llr_loader_t::load_sync(smem, params, blockIdx.x);

    //------------------------------------------------------------------
    // Perform iterations
    kernel_config_t::sched_t sched(smem,
                                   params,
                                   bgdesc,
                                   static_cast<int>(__cvta_generic_to_shared(smem)),
                                   threadIdx.x);
    for(int iter = 0; iter < params.max_iterations; ++iter)
    {
        sched.do_iteration();
        //thread0_dump_app(reinterpret_cast<__half2*>(smem), params.Z_var);
    }

    //------------------------------------------------------------------
    // Write hard output based on APP values
    //ldpc_dec_output_variable(params, reinterpret_cast<const kernel_config_t::app_buf_t*>(smem));
    ldpc_dec_output_variable_loop(params, reinterpret_cast<const kernel_config_t::app_buf_t*>(smem));
}

////////////////////////////////////////////////////////////////////////
// ldpc2_BG2_split_index_fp_x2_desc_dyn()
// Kernel for base graph 1 (legacy tensor interface)
extern "C"
__global__ __launch_bounds__(MAX_THREADS_PER_CTA, MIN_CTA_PER_SM)
void ldpc2_BG2_split_index_fp_x2_desc_dyn(LDPC_kernel_params params, app_loc_t<2>::bg_desc_t bgdesc)
{
    // Shared memory is allocated dynamically
    extern __shared__ char smem[];
    
    //------------------------------------------------------------------
    // Kernel configuration template
    typedef ldpc2_split_index_fp_x2_desc_dyn_kernel_config<2,                                          // BG
                                                           ldpc2::LDPC_kernel_params> kernel_config_t; // params struct

    //------------------------------------------------------------------
    // Load LLR data from global to shared memory
    kernel_config_t::llr_loader_t::load_sync(smem, params, blockIdx.x);

    //------------------------------------------------------------------
    // Perform iterations
    kernel_config_t::sched_t sched(smem,
                                   params,
                                   bgdesc,
                                   static_cast<int>(__cvta_generic_to_shared(smem)),
                                   threadIdx.x);
    for(int iter = 0; iter < params.max_iterations; ++iter)
    {
        sched.do_iteration();
    }

    //------------------------------------------------------------------
    // Write hard output based on APP values
    ldpc_dec_output_variable(params, reinterpret_cast<const kernel_config_t::app_buf_t*>(smem));
    // No loop needed for BG2 with Z>= 32
    //ldpc_dec_output_variable_loop(params, reinterpret_cast<const kernel_config_t::app_buf_t*>(smem));
}

////////////////////////////////////////////////////////////////////////
// ldpc2_BG1_split_index_fp_x2_desc_dyn_tb()
// Kernel for base graph 1 (transport block interface)
extern "C"
__global__ __launch_bounds__(MAX_THREADS_PER_CTA, MIN_CTA_PER_SM)
void ldpc2_BG1_split_index_fp_x2_desc_dyn_tb(cuphyLDPCDecodeDesc_t decodeDesc, app_loc_t<1>::bg_desc_t bgdesc)
{
    // Shared memory is allocated dynamically
    extern __shared__ char smem[];

    //------------------------------------------------------------------
    // Kernel configuration template
    typedef ldpc2_split_index_fp_x2_desc_dyn_kernel_config<1,                                            // BG
                                                           cuphyLDPCDecodeConfigDesc_t> kernel_config_t; // params struct
#if !LDPC_DECODE_USE_TB_SCAN
    //------------------------------------------------------------------
    // Load LLR data from global to shared memory
    kernel_config_t::llr_loader_t::load_sync(smem, decodeDesc, blockIdx.x);
#else
    tb_token tok = kernel_config_t::llr_loader_t::load_sync_token(smem,
                                                                  decodeDesc,
                                                                  blockIdx.x,
                                                                  get_token_addr<1>(decodeDesc, smem));
#endif

    //------------------------------------------------------------------
    // Perform iterations
    kernel_config_t::sched_t sched(smem,
                                   decodeDesc.config,
                                   bgdesc,
                                   static_cast<int>(__cvta_generic_to_shared(smem)),
                                   threadIdx.x);
    for(int iter = 0; iter < decodeDesc.config.max_iterations; ++iter)
    {
        sched.do_iteration();
    }

    //------------------------------------------------------------------
    // Write hard output based on APP values
    //ldpc_dec_output_variable(decodeDesc, reinterpret_cast<const kernel_config_t::app_buf_t*>(smem));
#if !LDPC_DECODE_USE_TB_SCAN
    ldpc_dec_output_variable_loop(decodeDesc, reinterpret_cast<const kernel_config_t::app_buf_t*>(smem));
#else
    ldpc_dec_output_variable_loop(decodeDesc,
                                  tok,
                                  reinterpret_cast<const kernel_config_t::app_buf_t*>(smem));
#endif
}

////////////////////////////////////////////////////////////////////////
// ldpc2_BG2_split_index_fp_x2_desc_dyn_tb()
// Kernel for base graph 2 (transport block interface)
extern "C"
__global__ __launch_bounds__(MAX_THREADS_PER_CTA, MIN_CTA_PER_SM)
void ldpc2_BG2_split_index_fp_x2_desc_dyn_tb(cuphyLDPCDecodeDesc_t decodeDesc, app_loc_t<2>::bg_desc_t bgdesc)
{
    // Shared memory is allocated dynamically
    extern __shared__ char smem[];
    
    //------------------------------------------------------------------
    // Kernel configuration template
    typedef ldpc2_split_index_fp_x2_desc_dyn_kernel_config<2,                                            // BG
                                                           cuphyLDPCDecodeConfigDesc_t> kernel_config_t; // params struct

#if !LDPC_DECODE_USE_TB_SCAN
    //------------------------------------------------------------------
    // Load LLR data from global to shared memory
    kernel_config_t::llr_loader_t::load_sync(smem, decodeDesc, blockIdx.x);
#else
    tb_token tok = kernel_config_t::llr_loader_t::load_sync_token(smem,
                                                                  decodeDesc,
                                                                  blockIdx.x,
                                                                  get_token_addr<2>(decodeDesc, smem));
#endif
    //------------------------------------------------------------------
    // Perform iterations
    kernel_config_t::sched_t sched(smem,
                                   decodeDesc.config,
                                   bgdesc,
                                   static_cast<int>(__cvta_generic_to_shared(smem)),
                                   threadIdx.x);
    for(int iter = 0; iter < decodeDesc.config.max_iterations; ++iter)
    {
        sched.do_iteration();
    }

    //------------------------------------------------------------------
    // Write hard output based on APP values
#if !LDPC_DECODE_USE_TB_SCAN
    ldpc_dec_output_variable(decodeDesc, reinterpret_cast<const kernel_config_t::app_buf_t*>(smem));
    // No loop needed for BG2 with Z>= 32
    //ldpc_dec_output_variable_loop(decodeDesc, reinterpret_cast<const kernel_config_t::app_buf_t*>(smem));
#else
    ldpc_dec_output_variable(decodeDesc,
                             tok,
                             reinterpret_cast<const kernel_config_t::app_buf_t*>(smem));
#endif
}

namespace ldpc2
{

////////////////////////////////////////////////////////////////////////
// split_index_fp_x2_desc_dyn::decode()
cuphyStatus_t split_index_fp_x2_desc_dyn::decode(ldpc::decoder&                     dec,
                                                 LDPC_output_t&                     tDst,
                                                 const_tensor_pair&                 tLLR,
                                                 const cuphyLDPCDecodeConfigDesc_t& config,
                                                 hipStream_t                       strm)
{
    DEBUG_PRINTF("ldpc::decode_ldpc2_split_index_fp_x2_desc_dyn()\n");
    //------------------------------------------------------------------
    cuphyDataType_t llrType = tLLR.first.get().type();
    const int       NUM_CW  = tLLR.first.get().layout().dimensions[1];
    //------------------------------------------------------------------
    dim3 grdDim(div_round_up(NUM_CW, 2));
    // We need to be mindful of the blockDim not being a multiple of 32.
    // The hard decision output writes 32-bit words. We may need to
    // revisit the output function to allow us to truncate the threads
    // that write to the next lowest multiple of 32, but that  may also
    // mean that we need to then have the output function LOOP.
    //dim3 blkDim(((config.Z + 31) / 32) * 32);
    dim3 blkDim(config.Z);

    //------------------------------------------------------------------
    // Initialize the kernel params struct
    LDPC_kernel_params params(config, tLLR, tDst);

    cuphyStatus_t s = CUPHY_STATUS_NOT_SUPPORTED;

    //------------------------------------------------------------------
    // Determine the dynamic amount of shared memory
    const uint32_t SHMEM_SIZE = get_shmem_required(config.BG,
                                                   config.num_parity_nodes,
                                                   config.Z);
    
    if(llrType == CUPHY_R_16F)
    {
        switch(config.BG)
        {
        case 1:
            {
                //------------------------------------------------------------------
                // Retrieve the base graph descriptor
                const app_loc_t<1>::bg_desc_t* bgdesc = app_loc_t<1>::get_bg_desc(params.Z);
                if(!bgdesc) break;
                
                DEBUG_PRINT_FUNC_MAX_BLOCKS(ldpc2_BG1_split_index_fp_x2_desc_dyn, blkDim, SHMEM_SIZE);
                
                //------------------------------------------------------------------
                // Launch the kernel
                ldpc2_BG1_split_index_fp_x2_desc_dyn<<<grdDim, blkDim, SHMEM_SIZE, strm>>>(params, *bgdesc);
                s = CUPHY_STATUS_SUCCESS;
            }
            break;
        case 2:
            {
                //------------------------------------------------------------------
                // Retrieve the base graph descriptor
                const app_loc_t<2>::bg_desc_t* bgdesc = app_loc_t<2>::get_bg_desc(params.Z);
                if(!bgdesc) break;

                DEBUG_PRINT_FUNC_MAX_BLOCKS(ldpc2_BG2_split_index_fp_x2_desc_dyn, blkDim, SHMEM_SIZE);
                
                //------------------------------------------------------------------
                // Launch the kernel
                ldpc2_BG2_split_index_fp_x2_desc_dyn<<<grdDim, blkDim, SHMEM_SIZE, strm>>>(params, *bgdesc);
                s = CUPHY_STATUS_SUCCESS;
            }
            break;
        default:
            break;
        }
    }

    if(CUPHY_STATUS_SUCCESS != s)
    {
        return s;
    }

#if CUPHY_DEBUG
    hipDeviceSynchronize();
#endif
    hipError_t e = hipGetLastError();
    DEBUG_PRINTF("CUDA STATUS (%s:%i): %s\n", __FILE__, __LINE__, hipGetErrorString(e));
    return (e == hipSuccess) ? CUPHY_STATUS_SUCCESS : CUPHY_STATUS_INTERNAL_ERROR;
}

////////////////////////////////////////////////////////////////////////
// split_index_fp_x2_desc_dyn::decode_tb()
cuphyStatus_t split_index_fp_x2_desc_dyn::decode_tb(ldpc::decoder&               dec,
                                                    const cuphyLDPCDecodeDesc_t& decodeDesc,
                                                    hipStream_t                 strm)
{
    DEBUG_PRINTF("ldpc2::split_index_fp_x2_desc_dyn::decode_tb()\n");
    
    cuphyStatus_t s = CUPHY_STATUS_NOT_SUPPORTED;

    if(decodeDesc.config.llr_type == CUPHY_R_16F)
    {
        // We need to be mindful of the blockDim not being a multiple of 32.
        // The hard decision output writes 32-bit words. We may need to
        // revisit the output function to allow us to truncate the threads
        // that write to the next lowest multiple of 32, but that  may also
        // mean that we need to then have the output function LOOP.
        //dim3 blkDim(((config.Z + 31) / 32) * 32);
        dim3 blkDim(decodeDesc.config.Z);
        
        //------------------------------------------------------------------
        // Launch a CTA for each codeword pair. Note that the number of CTAs
        // may be more than the total number of codewords divided by 2 -
        // there may be transport blocks with odd numbers of codewords.
        dim3 grdDim(ldpc::decoder::get_total_num_codeword_pairs(decodeDesc));

        //------------------------------------------------------------------
        // Determine the dynamic amount of shared memory
        const uint32_t SHMEM_SIZE = get_shmem_required(decodeDesc.config.BG,
                                                       decodeDesc.config.num_parity_nodes,
                                                       decodeDesc.config.Z);
        switch(decodeDesc.config.BG)
        {
        case 1:
            {
                //------------------------------------------------------------------
                // Retrieve the base graph descriptor
                const app_loc_t<1>::bg_desc_t* bgdesc = app_loc_t<1>::get_bg_desc(decodeDesc.config.Z);
                if(!bgdesc) break;
                
                DEBUG_PRINT_FUNC_MAX_BLOCKS(ldpc2_BG1_split_index_fp_x2_desc_dyn_tb, blkDim, SHMEM_SIZE);

                //------------------------------------------------------------------
                // Launch the kernel
               ldpc2_BG1_split_index_fp_x2_desc_dyn_tb<<<grdDim, blkDim, SHMEM_SIZE, strm>>>(decodeDesc, *bgdesc);
                s = CUPHY_STATUS_SUCCESS;
            }
            break;
        case 2:
            {
                //------------------------------------------------------------------
                // Retrieve the base graph descriptor
                const app_loc_t<2>::bg_desc_t* bgdesc = app_loc_t<2>::get_bg_desc(decodeDesc.config.Z);
                if(!bgdesc) break;
                
                DEBUG_PRINT_FUNC_MAX_BLOCKS(ldpc2_BG2_split_index_fp_x2_desc_dyn_tb, blkDim, SHMEM_SIZE);
                
                //------------------------------------------------------------------
                // Launch the kernel
                ldpc2_BG2_split_index_fp_x2_desc_dyn_tb<<<grdDim, blkDim, SHMEM_SIZE, strm>>>(decodeDesc, *bgdesc);
                s = CUPHY_STATUS_SUCCESS;
            }
            break;
        default:
            break;
        }
    }
    if(CUPHY_STATUS_SUCCESS != s)
    {
        return s;
    }

#if CUPHY_DEBUG
    hipDeviceSynchronize();
#endif
    hipError_t e = hipGetLastError();
    DEBUG_PRINTF("CUDA STATUS (%s:%i): %s\n", __FILE__, __LINE__, hipGetErrorString(e));
    return (e == hipSuccess) ? CUPHY_STATUS_SUCCESS : CUPHY_STATUS_INTERNAL_ERROR;
}
////////////////////////////////////////////////////////////////////////
// split_index_fp_x2_desc_dyn::get_workspace_size()
std::pair<bool, size_t> split_index_fp_x2_desc_dyn::get_workspace_size(const ldpc::decoder&               dec,
                                                                       const cuphyLDPCDecodeConfigDesc_t& config,
                                                                       int                                num_cw)
{
    return std::pair<bool, size_t>(true, 0);
}

////////////////////////////////////////////////////////////////////////
// split_index_fp_x2_desc_dyn::split_index_fp_x2_desc_dyn()
split_index_fp_x2_desc_dyn::split_index_fp_x2_desc_dyn(ldpc::decoder& dec)
{
    //------------------------------------------------------------------
    // Determine the maximum amount of shared memory that could be used
    // by a kernel
    const int MAX_BG1_SHMEM_SIZE = static_cast<int>(get_shmem_required(1,                             // BG
                                                                       max_num_parity<1>::value,      // max parity nodes
                                                                       CUPHY_LDPC_MAX_LIFTING_SIZE)); // lifting size
    const int MAX_BG2_SHMEM_SIZE = static_cast<int>(get_shmem_required(2,                             // BG
                                                                       max_num_parity<2>::value,      // max parity nodes
                                                                       CUPHY_LDPC_MAX_LIFTING_SIZE)); // lifting size
    //------------------------------------------------------------------
    // Maximum shared memory supported by the device
    const int MAX_SHMEM = dec.max_shmem_per_block_optin();

    //------------------------------------------------------------------
    // For each kernel, set the maximum dynamic shared memory size
    typedef std::pair<const void*, int> func_attr_t;
    std::array<func_attr_t, 4> func_attrs =
    {
        func_attr_t((const void*)ldpc2_BG1_split_index_fp_x2_desc_dyn,    std::min(MAX_BG1_SHMEM_SIZE, MAX_SHMEM)),
        func_attr_t((const void*)ldpc2_BG2_split_index_fp_x2_desc_dyn,    std::min(MAX_BG2_SHMEM_SIZE, MAX_SHMEM)),
        func_attr_t((const void*)ldpc2_BG1_split_index_fp_x2_desc_dyn_tb, std::min(MAX_BG1_SHMEM_SIZE, MAX_SHMEM)),
        func_attr_t((const void*)ldpc2_BG2_split_index_fp_x2_desc_dyn_tb, std::min(MAX_BG2_SHMEM_SIZE, MAX_SHMEM))
    };
    for(func_attr_t f_a : func_attrs)
    {
        hipError_t e = hipFuncSetAttribute(reinterpret_cast<const void*>(f_a.first),
                                             hipFuncAttributeMaxDynamicSharedMemorySize,
                                             f_a.second);
        if(hipSuccess != e)
        {
            throw cuphy_i::cuda_exception(e);
        }
    }
    //------------------------------------------------------------------
    DEBUG_PRINT_FUNC_ATTRIBUTES(ldpc2_BG1_split_index_fp_x2_desc_dyn);
    DEBUG_PRINT_FUNC_ATTRIBUTES(ldpc2_BG2_split_index_fp_x2_desc_dyn);
    DEBUG_PRINT_FUNC_ATTRIBUTES(ldpc2_BG1_split_index_fp_x2_desc_dyn_tb);
    DEBUG_PRINT_FUNC_ATTRIBUTES(ldpc2_BG2_split_index_fp_x2_desc_dyn_tb);
}

////////////////////////////////////////////////////////////////////////
// split_index_fp_x2_desc_dyn::can_decode_config()
bool split_index_fp_x2_desc_dyn::can_decode_config(const ldpc::decoder&               dec,
                                                   const cuphyLDPCDecodeConfigDesc_t& cfg)
{
    // Compare shared memory requirements to device maximum, as well as
    // the maximum that the kernel was compiled for.
    
    // Maximum number of parity nodes, as limited by compilation, to
    // limit register usage.
    const uint32_t MAX_NUM_PARITY = (1 == cfg.BG) ? max_num_parity<1>::value : max_num_parity<2>::value;
    // Calculate required shared memory
    const uint32_t SHMEM_BYTES    = get_shmem_required(cfg.BG,
                                                       cfg.num_parity_nodes,
                                                       cfg.Z);
    return (cfg.num_parity_nodes <= MAX_NUM_PARITY) &&
           (SHMEM_BYTES <= dec.max_shmem_per_block_optin());
}

////////////////////////////////////////////////////////////////////////
// split_index_fp_x2_desc_dyn::get_launch_config()
cuphyStatus_t split_index_fp_x2_desc_dyn::get_launch_config(const ldpc::decoder&           dec,
                                                            cuphyLDPCDecodeLaunchConfig_t& launchConfig)
{
    const int Z                = launchConfig.decode_desc.config.Z;
    const int BG               = launchConfig.decode_desc.config.BG;
    const int NUM_PARITY_NODES = launchConfig.decode_desc.config.num_parity_nodes;
    const int MAX_PARITY_NODES = (1 == BG)                  ?
                                 max_parity_nodes<1>::value :
                                 max_parity_nodes<2>::value;
    const int NUM_VAR_NODES    = ldpc::decoder::get_num_variable_nodes(BG,
                                                                       NUM_PARITY_NODES);
    //------------------------------------------------------------------
    // Validate input arguments
    if((Z < 2)                              ||
       (Z > CUPHY_LDPC_MAX_LIFTING_SIZE)    ||
       (NUM_PARITY_NODES < 4)               ||
       (NUM_PARITY_NODES > MAX_PARITY_NODES))
    {
        return CUPHY_STATUS_UNSUPPORTED_CONFIG;
    }
    //------------------------------------------------------------------
    // Set up launch geometry and the kernel function (driver)
    #if CUDART_VERSION >= 11000
    launchConfig.kernel_node_params_driver.blockDimX = Z;
    launchConfig.kernel_node_params_driver.blockDimY = 1;
    launchConfig.kernel_node_params_driver.blockDimZ = 1;

    launchConfig.kernel_node_params_driver.gridDimX = ldpc::decoder::get_total_num_codeword_pairs(launchConfig.decode_desc);
    launchConfig.kernel_node_params_driver.gridDimY = 1;
    launchConfig.kernel_node_params_driver.gridDimZ = 1;

    launchConfig.kernel_node_params_driver.extra          = nullptr;
    launchConfig.kernel_node_params_driver.kernelParams   = launchConfig.kernel_args;

    const uint32_t SHMEM_SIZE = get_shmem_required(launchConfig.decode_desc.config.BG,
                                                   launchConfig.decode_desc.config.num_parity_nodes,
                                                   launchConfig.decode_desc.config.Z);
    launchConfig.kernel_node_params_driver.sharedMemBytes = SHMEM_SIZE;

    hipFunction_t deviceFunction;
    hipError_t    e = (BG == 1) ? hipGetFuncBySymbol(&deviceFunction, (void*)ldpc2_BG1_split_index_fp_x2_desc_dyn_tb) : 
                                   hipGetFuncBySymbol(&deviceFunction, (void*)ldpc2_BG2_split_index_fp_x2_desc_dyn_tb);
    if (e != hipSuccess) 
    {
        return CUPHY_STATUS_INTERNAL_ERROR;
    }
    launchConfig.kernel_node_params_driver.func = static_cast<hipFunction_t>(deviceFunction);
    #endif
    //------------------------------------------------------------------
    // Set kernel arguments:
    // arg 0: decode descriptor
    launchConfig.kernel_args[0] = &launchConfig.decode_desc;
    // arg 1: base graph descriptor
    if(1 == BG)
    {
        const app_loc_t<1>::bg_desc_t* bgdesc = app_loc_t<1>::get_bg_desc(Z);
        launchConfig.kernel_args[1] = const_cast<void*>(reinterpret_cast<const void*>(bgdesc));
    }
    else
    {
        const app_loc_t<2>::bg_desc_t* bgdesc = app_loc_t<2>::get_bg_desc(Z);
        launchConfig.kernel_args[1] = const_cast<void*>(reinterpret_cast<const void*>(bgdesc));
    }
    return CUPHY_STATUS_SUCCESS;
}

} // namespace ldpc2
