#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 */
 
#include "ofdmMod.cuh"
#include <cufftdx.hpp>

using namespace ofdm_modulate;

/**
 * @brief main kernel for ofdm modulation
 * 
 * @tparam FFT FFT configurations, see cuFFTdx documents for detals
 * @tparam Tscalar scalar template, must match with Tcomplex
 * @tparam Tcomplex comlex template, must match with Tscalar
 * @param ofdmModdynDescpr ofdm demodulation dynamic descriptor
 * 
 * @param GridDim m_ofdmDeModdynDescprCpu -> N_txLayer, cuphyCarrierPrms -> N_symble_slot / OFDM_FFTs_PER_BLOCK_CONST_, 1
 * @param BlockDim defuallt set by cuFFTdx
 */
template<typename FFT, typename Tscalar, typename Tcomplex>
__launch_bounds__(FFT::max_threads_per_block)
static __global__ void ofdmMod_ifft_kernel(ofdmModDynDescr_t<Tscalar, Tcomplex> * ofdmModdynDescpr)
{
    using namespace cufftdx;
    // Registers
    hipComplex thread_data[FFT::storage_size];
    Tcomplex * freqDataIn = ofdmModdynDescpr -> freqDataIn;
    Tcomplex * timeDataOut = ofdmModdynDescpr -> timeDataOut;
    uint N_sc_over_2 = ofdmModdynDescpr -> N_sc >> 1; // divide by 2
    uint N_IFFT = ofdmModdynDescpr -> N_IFFT;
    // Local batch id of this FFT in CUDA block, in range [0; FFT::ffts_per_block)
    const unsigned int local_fft_id = threadIdx.y;
    // Global batch id of this FFT in CUDA grid is equal to number of batches per CUDA block (ffts_per_block)
    // times CUDA block id, plus local batch id.
    const unsigned int global_fft_id = (blockIdx.x * gridDim.y + blockIdx.y) * FFT::ffts_per_block + local_fft_id;
    // blockIdx.x = N_layer, blockIdx.y = N_symble_slot (14 or 12) / OFDM_FFTs_PER_BLOCK_CONST_ 

    /*-------------------   load data into shared memoery for IFFT-------------------*/
    // Load freq data from global memory to registers
    const unsigned int freq_offset = ofdmModdynDescpr -> N_sc * global_fft_id;
    const unsigned int CP_current = ofdmModdynDescpr -> cpInfo[blockIdx.y * FFT::ffts_per_block + threadIdx.y]; // CP length for current OFDM symbol
    const unsigned int CP_offset  = ofdmModdynDescpr -> cpInfo[blockIdx.y * FFT::ffts_per_block + threadIdx.y + (ofdmModdynDescpr -> N_symble_slot)]; // CP_offset in this layer
    const unsigned int time_offset = cufftdx::size_of<FFT>::value * global_fft_id + CP_offset + (ofdmModdynDescpr -> cpInfo[(ofdmModdynDescpr -> N_symble_slot) * 2 - 1]) * blockIdx.x; // FFT size + CP offset in this layer + CP offset in previous layers
    const unsigned int stride = FFT::stride;
    unsigned int       index  = threadIdx.x;

    for (unsigned int i = 0; i < FFT::elements_per_thread; i++) 
    {
        // Make sure not to go out-of-bounds
        if ((i * stride + threadIdx.x) < cufftdx::size_of<FFT>::value) 
        {
            #ifdef USE_MEMOERY_FFT_SHIFT_ // perform ifftshift first
            if(index < N_sc_over_2) // CUPHY_N_TONES_PER_PRB = 12
            {
                thread_data[i].x = freqDataIn[index + freq_offset + N_sc_over_2].x;
                thread_data[i].y = freqDataIn[index + freq_offset + N_sc_over_2].y; // first half
            }
            else if( index >= (N_IFFT - N_sc_over_2))
            {
                thread_data[i].x = freqDataIn[index + freq_offset - N_IFFT + N_sc_over_2].x;
                thread_data[i].y = freqDataIn[index + freq_offset - N_IFFT + N_sc_over_2].y; // second half
            }
            else // zero otherwise
            {
                thread_data[i].x = 0.0f; 
                thread_data[i].y = 0.0f; 
            }
            #else // no ifftshift 
            if(index >= ((N_IFFT >> 1) - N_sc_over_2) && index < ((N_IFFT >> 1) + N_sc_over_2) ) // Middl part
            {
                thread_data[i] = freqDataIn[index + freq_offset - ((N_IFFT >> 1) - N_sc_over_2)]; 
            }
            else // zero otherwise
            {
                thread_data[i].x = 0.0f; 
                thread_data[i].y = 0.0f; 
            }
            #endif

            index += stride;
        }
        // printf("IFFT in: global_fft_id=%d, threadIdx.x=%d, threadIdx.y=%d, i=%d, (i * stride + threadIdx.x) = %d: thread_data[i].x = %f, thread_data[i].y = %f \n", global_fft_id, threadIdx.x, threadIdx.y, i, (i * stride + threadIdx.x), float(thread_data[i].x), float(thread_data[i].y));
    }

    // FFT::shared_memory_size bytes of shared memory
    using complex_type = typename FFT::value_type;
    extern __shared__ complex_type shared_mem[];

    // Execute IFFT
    FFT().execute(thread_data, shared_mem);

    /*-------------------   Add CPs  -------------------*/
    index = time_offset + threadIdx.x;
#pragma unroll
    for (unsigned int i = 0; i < FFT::elements_per_thread; i++) 
    {
        if ((i * stride + threadIdx.x) < cufftdx::size_of<FFT>::value) 
        {
            // normalization
            thread_data[i].x = thread_data[i].x * ofdmModdynDescpr -> sqrt_N_IFFT_inverse; // normalize by sqrt(N_IFFT)
            thread_data[i].y = thread_data[i].y * ofdmModdynDescpr -> sqrt_N_IFFT_inverse; // normalize by sqrt(N_IFFT)
            
            // real part
            #ifdef USE_MEMOERY_FFT_SHIFT_ // no change due to ifftshift first
            timeDataOut[index].x = thread_data[i].x;
            timeDataOut[index].y = thread_data[i].y;

            if((i * stride + threadIdx.x) >= N_IFFT - CP_current) // copy CP
            {
                timeDataOut[index - N_IFFT].x = thread_data[i].x;
                timeDataOut[index - N_IFFT].y = thread_data[i].y;
            }
            #else // times 1 or -1 to real part due to no ifftshift
            if(index & 1) // last bit is 1
            {
                timeDataOut[index].x = - thread_data[i].x;   
                timeDataOut[index].y = - thread_data[i].y;
                
                if((i * stride + threadIdx.x) >= N_IFFT - CP_current) // copy CP
                {
                    timeDataOut[index - N_IFFT].x = - thread_data[i].x;
                    timeDataOut[index - N_IFFT].y = - thread_data[i].y;
                }
            }
            else
            {
                timeDataOut[index].x = thread_data[i].x; 
                timeDataOut[index].y = thread_data[i].y; 

                if((i * stride + threadIdx.x) >= N_IFFT - CP_current) // copy CP
                {
                    timeDataOut[index - N_IFFT].x = thread_data[i].x; 
                    timeDataOut[index - N_IFFT].y = thread_data[i].y; 
                }
            }
            #endif
            index += stride;
        }
        // printf("IFFT out: global_fft_id=%d, threadIdx.x=%d,  threadIdx.y=%d, i=%d, (i * stride + threadIdx.x) = %d: thread_data[i].x = %f, thread_data[i].y = %f \n", global_fft_id, threadIdx.x, threadIdx.y, i, (i * stride + threadIdx.x), float(thread_data[i].x), float(thread_data[i].y));
    }
}


/**
 * @brief Apply windowing using raised cosine if needed
 * 
 * @param ofdmModdynDescpr ofdm dynamic descriptor
 * @todo not tested yet
 */
template<typename Tscalar, typename Tcomplex>
static __global__ void applyWindow(ofdmModDynDescr_t<Tscalar, Tcomplex> * ofdmModdynDescpr)
{
    // blockIdx.x = layerIdx, threadIdx.y = ofdmIdx, threadIdx.x = [0 ~ ofdmModdynDescpr -> ofdmWindowLen -1];

    uint N_IFFT = ofdmModdynDescpr -> N_IFFT;
    const unsigned int ofdmSymId = threadIdx.y;
    const unsigned int CP_current = ofdmModdynDescpr -> cpInfo[ofdmSymId]; // CP length for current OFDM symbol
    const unsigned int CP_offset  = ofdmModdynDescpr -> cpInfo[ofdmSymId + (ofdmModdynDescpr -> N_symble_slot)]; // CP_offset in this layer
    const unsigned int block_offset = N_IFFT*ofdmSymId + CP_offset + (N_IFFT * (ofdmModdynDescpr -> N_symble_slot) + ofdmModdynDescpr -> cpInfo[(ofdmModdynDescpr -> N_symble_slot) * 2 - 1]) * blockIdx.x; // FFT size + CP offset in this layer + CP offset in previous layers

    /*---------    Apply windowing  ------------ */
    if(ofdmModdynDescpr -> ofdmWindowLen > 2) // need to apply CP
    {
        uint index;
        Tcomplex temp_windowing;
        // get suffix 
        if(ofdmSymId < (ofdmModdynDescpr -> N_symble_slot - 1)) 
        {
            index = block_offset + threadIdx.x; // FFT start of symbol (ofdmSymId-1) for suffix
            temp_windowing.x =  ofdmModdynDescpr -> ofdmWindowCoe[threadIdx.x] * (ofdmModdynDescpr -> timeDataOut[index]).x;
            temp_windowing.y =  ofdmModdynDescpr -> ofdmWindowCoe[threadIdx.x] * (ofdmModdynDescpr -> timeDataOut[index]).y;

            index = index + N_IFFT;
        }
        else // i.e., ofdmSymId = ofdmModdynDescpr -> N_symble_slot - 1
        // only apply prefix window to the first symbol, Fix me if multiple slots simulated together
        {
            (ofdmModdynDescpr -> timeDataOut[threadIdx.x]).x = ofdmModdynDescpr -> ofdmWindowCoe[ofdmModdynDescpr -> ofdmWindowLen - 1 - threadIdx.x] * (ofdmModdynDescpr -> timeDataOut[index]).x; 
            (ofdmModdynDescpr -> timeDataOut[threadIdx.x]).y = ofdmModdynDescpr -> ofdmWindowCoe[ofdmModdynDescpr -> ofdmWindowLen - 1 - threadIdx.x] * (ofdmModdynDescpr -> timeDataOut[index]).y; 
        }
    }
}

template<typename Tscalar, typename Tcomplex>
using ifftKernelHandle = void (*)(ofdmModDynDescr_t<Tscalar, Tcomplex> * ofdmModdynDescpr);

// Choose IFFT kernel
template<typename Tscalar, typename Tcomplex, unsigned int FftSize, unsigned int Arch>
ifftKernelHandle<Tscalar, Tcomplex> ofdmMod_get_ifft_param(dim3& block_dim, uint& shared_memory_size) 
{ 
    using namespace cufftdx;

    // use predefined numbers
    using FFT = decltype(Size<FftSize>() + Precision<float>() + Type<fft_type::c2c>()
                                + Direction<fft_direction::inverse>()
                                + FFTsPerBlock<OFDM_FFTs_PER_BLOCK_CONST_>() // + ElementsPerThread<8>()
                                + SM<Arch>() + Block());
    
    // use cuFFTdx configurations
    // Base of the FFT description
    // using FFT_base = decltype(Size<FftSize>() + Precision<Tscalar>() + Type<fft_type::c2c>()
    //                             + Direction<fft_direction::inverse>()
    //                             /* Notice lack of ElementsPerThread and FFTsPerBlock operators */
    //                             + SM<Arch>() + Block());
    // // FFT description with suggested FFTs per CUDA block for the default (optimal) elements per thread
    // using FFT = decltype(FFT_base() + FFTsPerBlock<1>());

    block_dim = FFT::block_dim;
    shared_memory_size = FFT::shared_memory_size;

    return ofdmMod_ifft_kernel<FFT, Tscalar, Tcomplex>;
 }

 /**
  * @brief get ifft kernel handles
  * 
  * @param Nifft IFFT size
  * @param cudaDeviceArch GPU device arch
  * @param block_dim auto config by cuFFTdx
  * @param shared_memory_size auto config by cuFFTdx
  * @return fftKernelHandle<Tscalar, Tcomplex>
  * 
  * @note To conserve memeory, only selected IFFT size and cudaDeviceArch are added. If your Nifft and cudaDeviceArch are not in the below list, please add them and retry the build
  */
template<typename Tscalar, typename Tcomplex>
ifftKernelHandle<Tscalar, Tcomplex> ofdmMod_get_ifft_param(const int Nifft, unsigned int cudaDeviceArch, dim3& block_dim, uint& shared_memory_size) 
{ 
    // current only support cudaDeviceArch = 800
    switch(Nifft) 
    {
        case 256:
            return ofdmMod_get_ifft_param<Tscalar, Tcomplex,  512, 800>(block_dim, shared_memory_size);
            break;
        case 512:
            return ofdmMod_get_ifft_param<Tscalar, Tcomplex,  512, 800>(block_dim, shared_memory_size);
            break;
        case 1024:
            return ofdmMod_get_ifft_param<Tscalar, Tcomplex,  1024, 800>(block_dim, shared_memory_size);
            break;
        case 2048:
            return ofdmMod_get_ifft_param<Tscalar, Tcomplex,  2048, 800>(block_dim, shared_memory_size);
            break;
        case 4096:
            return ofdmMod_get_ifft_param<Tscalar, Tcomplex,  4096, 800>(block_dim, shared_memory_size);
            break;
        default:
            printf("Unsupported IFFT length %d or cudaDeviceArch %d in OFDM modulation, please add your Nifft or cudaDeviceArch into ofdmMod_get_ifft_param and retry\n", Nifft, cudaDeviceArch); 
            assert(false);
            return nullptr;
    }
    return nullptr;
}

template <typename Tscalar, typename Tcomplex> 
ofdmModulate<Tscalar, Tcomplex>::ofdmModulate(cuphyCarrierPrms_t * cuphyCarrierPrms, Tcomplex * freqDataIn, hipStream_t strm)
{
    uint mu = cuphyCarrierPrms -> mu;
    uint N_symble_slot = cuphyCarrierPrms -> N_symble_slot;
    //m_N_IFFT = cuphyCarrierPrms -> N_IFFT;
    m_ofdmModdynDescprCpu = new ofdmModDynDescr_t<Tscalar, Tcomplex>;
    m_ofdmModdynDescprCpu -> N_IFFT = cuphyCarrierPrms -> N_FFT;
    m_ofdmModdynDescprCpu -> sqrt_N_IFFT_inverse = 1.0f/sqrt(cuphyCarrierPrms -> N_FFT);
    m_ofdmModdynDescprCpu -> N_sc = cuphyCarrierPrms -> N_sc;
    m_ofdmModdynDescprCpu -> N_txLayer = cuphyCarrierPrms -> N_txLayer;
    m_ofdmModdynDescprCpu -> mu = mu;
    m_ofdmModdynDescprCpu -> N_symble_slot = N_symble_slot;
    uint symbol0IdxPerSubFrame = (cuphyCarrierPrms -> id_slot) * N_symble_slot;

    /* ----------------  CP info ---------------------- */
    uint16_t cpInfoLen = (N_symble_slot << 1);
    m_cpInfoCpu = new uint16_t[cpInfoLen]; // [CP info, accumCP] 
    hipMalloc((void**)&m_cpInfoGpu, sizeof(uint16_t) * cpInfoLen);
    // calculate CP length
    float T_c_over_T_samp = float(cuphyCarrierPrms->f_samp)/float(cuphyCarrierPrms->f_c);
    if(cuphyCarrierPrms -> cpType == 0) // normal CP length
    {
        uint16_t lenCP0 = (((144 >> mu) + 16) << (cuphyCarrierPrms -> kappa_bits))*T_c_over_T_samp; //(144+16)/2048*Nifft;
        uint16_t lenCP1 = ((144 >> mu) << (cuphyCarrierPrms -> kappa_bits))*T_c_over_T_samp; // 144/2048*Nifft;
        
        for(uint8_t symbolIdx = 0; symbolIdx < N_symble_slot; symbolIdx++)
        {
            m_cpInfoCpu[symbolIdx] = lenCP1;
        }

        if(mu == 0 && symbol0IdxPerSubFrame == 0)
        {
            m_cpInfoCpu[0] = lenCP0;
            m_cpInfoCpu[7] = lenCP0;
        } // check number of OFDM symbols per layer
        else if(mu != 0 && (symbol0IdxPerSubFrame == 0 || symbol0IdxPerSubFrame == (7 << mu)))
        {
            m_cpInfoCpu[0] = lenCP0;
        }
    }
    else
    {
        if(mu != 2)
        {
            printf("Error! Extended CP only applible in numerology 2! \n");
            exit(1);
        }
        uint lenCP1 = ((512 >> mu) << (cuphyCarrierPrms -> kappa_bits))*T_c_over_T_samp;
        for(uint8_t symbolIdx = 0; symbolIdx < N_symble_slot; symbolIdx++)
        {
            m_cpInfoCpu[symbolIdx] = lenCP1;
        }
    }
    
    // calculate assumualte CP
    m_cpInfoCpu[N_symble_slot] = m_cpInfoCpu[0];
    for(uint8_t symbolIdx = 1; symbolIdx < N_symble_slot; symbolIdx++)
    {
        m_cpInfoCpu[symbolIdx + N_symble_slot] = m_cpInfoCpu[symbolIdx] + m_cpInfoCpu[symbolIdx + N_symble_slot -1];
    }
    // copy CP info
    m_ofdmModdynDescprCpu -> cpInfo = m_cpInfoGpu;
    hipMemcpy(m_cpInfoGpu, m_cpInfoCpu, sizeof(uint16_t) * cpInfoLen, hipMemcpyHostToDevice);
    
    m_ofdmModdynDescprCpu -> freqDataIn = freqDataIn;
    m_timeDataLen = ((m_ofdmModdynDescprCpu -> N_IFFT) * (cuphyCarrierPrms -> N_symble_slot) + m_cpInfoCpu[cpInfoLen - 1]) * (m_ofdmModdynDescprCpu -> N_txLayer);
    hipMalloc((void**)&(m_ofdmModdynDescprCpu -> timeDataOut), sizeof(Tcomplex)*m_timeDataLen);

    /* ----------------  kernel launch config ---------------------- */
    // copy dynamic descriptor to GPU
    hipMalloc((void**)&m_ofdmModdynDescprGpu, sizeof(ofdmModDynDescr_t<Tscalar, Tcomplex>));
    hipMemcpy(m_ofdmModdynDescprGpu, m_ofdmModdynDescprCpu, sizeof(ofdmModDynDescr_t<Tscalar, Tcomplex>), hipMemcpyHostToDevice);

    // OFDM modulation kernel launch config
    m_pOfdmModCfg = new launchCfg_t;

    // set up kernel
    using namespace cufftdx;
    uint shared_memory_size = 0;
    dim3 block_dim;
    const uint cudaDeviceArch = get_cuda_device_arch();
    auto kernelPtr = ofdmMod_get_ifft_param<Tscalar, Tcomplex>( m_ofdmModdynDescprCpu -> N_IFFT, cudaDeviceArch, block_dim, shared_memory_size);
    m_pOfdmModCfg->kernelArgs[0] = &m_ofdmModdynDescprGpu;

    hipKernelNodeParams& ofdmModKernelNodeParams = m_pOfdmModCfg->kernelNodeParamsDriver;
    CUDA_CHECK(hipGetFuncBySymbol(&ofdmModKernelNodeParams.func, reinterpret_cast<void*>(kernelPtr)));
    // ofdmModKernelNodeParams.func = kernelPtr;
    ofdmModKernelNodeParams.blockDimX = block_dim.x;
    ofdmModKernelNodeParams.blockDimY = block_dim.y;
    ofdmModKernelNodeParams.blockDimZ = block_dim.z;

    ofdmModKernelNodeParams.gridDimX = m_ofdmModdynDescprCpu -> N_txLayer;
    ofdmModKernelNodeParams.gridDimY = cuphyCarrierPrms -> N_symble_slot / OFDM_FFTs_PER_BLOCK_CONST_;
    ofdmModKernelNodeParams.gridDimZ = 1;
    ofdmModKernelNodeParams.sharedMemBytes = shared_memory_size;
    ofdmModKernelNodeParams.kernelParams = &(m_pOfdmModCfg->kernelArgs[0]);
    ofdmModKernelNodeParams.extra = NULL;

    /* ----------------  OFDM windowning ---------------------- */
    // calculate window: OFDM Raised Cosine Window
    // NOT USED FOR NOW
    uint ofdmWindowLen = cuphyCarrierPrms -> ofdmWindowLen;
    m_ofdmModdynDescprCpu -> ofdmWindowLen = ofdmWindowLen;
    float rolloffFactor = cuphyCarrierPrms -> rolloffFactor;
    if(ofdmWindowLen > 3)
    {
        m_ofdmWindowCpu = new Tscalar[ofdmWindowLen];
        hipMalloc((void**)&m_ofdmWindowGpu, sizeof(Tscalar)*ofdmWindowLen);
        m_ofdmWindowCpu[0] = 1;
        m_ofdmWindowCpu[ofdmWindowLen - 1] = 0;
        float step = 1.0f/(ofdmWindowLen-1);
        for(int windowIdx=1; windowIdx < ofdmWindowLen-1; windowIdx++)
        {
            float t_over_T = windowIdx * step;
            m_ofdmWindowCpu[windowIdx] = sin(M_PI * t_over_T)/(M_PI * t_over_T) * cos(M_PI * t_over_T * rolloffFactor) / (1 - 4 * rolloffFactor * rolloffFactor * t_over_T * t_over_T);
        }
        hipMemcpy(m_ofdmWindowGpu, m_ofdmWindowCpu, sizeof(Tscalar)*ofdmWindowLen, hipMemcpyHostToDevice);
        m_ofdmModdynDescprCpu -> ofdmWindowCoe = m_ofdmWindowGpu;

        // apply windowing lauch config
        m_pWindowCfg = new launchCfg_t;
        hipKernelNodeParams& windowKernelNodeParams = m_pWindowCfg->kernelNodeParamsDriver;
        m_pOfdmModCfg->kernelArgs[0] = &m_ofdmModdynDescprGpu;
        CUDA_CHECK(hipGetFuncBySymbol(&windowKernelNodeParams.func, reinterpret_cast<void*>(applyWindow<Tscalar, Tcomplex>)));
        // windowKernelNodeParams.func = kernelPtr;
        windowKernelNodeParams.blockDimX = m_ofdmModdynDescprCpu -> ofdmWindowLen;
        windowKernelNodeParams.blockDimY = m_ofdmModdynDescprCpu -> N_symble_slot;
        windowKernelNodeParams.blockDimZ = 1;

        windowKernelNodeParams.gridDimX = m_ofdmModdynDescprCpu -> N_txLayer;
        windowKernelNodeParams.gridDimY = 1;
        windowKernelNodeParams.gridDimZ = 1;
        windowKernelNodeParams.sharedMemBytes = 0;
        windowKernelNodeParams.kernelParams = &(m_pOfdmModCfg->kernelArgs[0]);
        windowKernelNodeParams.extra = NULL;
    }
    else
    {
        m_ofdmModdynDescprCpu -> ofdmWindowCoe = nullptr;
        m_pWindowCfg = nullptr;
    }

    // pre load IFFT kernel to avoid first run timing
    run(strm);
}

template <typename Tscalar, typename Tcomplex> 
ofdmModulate<Tscalar, Tcomplex>::~ofdmModulate()
{
    delete[] m_cpInfoCpu;
    hipFree(m_cpInfoGpu);

    if(m_ofdmModdynDescprCpu -> ofdmWindowLen)
    {
        delete[] m_ofdmWindowCpu;
        hipFree(m_ofdmWindowGpu);
        delete m_pWindowCfg;
    }
    hipFree(m_ofdmModdynDescprGpu);
    hipFree(m_ofdmModdynDescprCpu -> timeDataOut);
    delete m_ofdmModdynDescprCpu;
    delete m_pOfdmModCfg;
}

template <typename Tscalar, typename Tcomplex> 
void ofdmModulate<Tscalar, Tcomplex>::run(hipStream_t strm)
{
    // launch ofdm modulation kernel
    const hipKernelNodeParams& ofdmModKernelNodeParams = m_pOfdmModCfg->kernelNodeParamsDriver;
    hipError_t runStatus = hipModuleLaunchKernel(ofdmModKernelNodeParams.func,
                                        ofdmModKernelNodeParams.gridDimX,
                                        ofdmModKernelNodeParams.gridDimY, 
                                        ofdmModKernelNodeParams.gridDimZ,
                                        ofdmModKernelNodeParams.blockDimX, 
                                        ofdmModKernelNodeParams.blockDimY, 
                                        ofdmModKernelNodeParams.blockDimZ,
                                        ofdmModKernelNodeParams.sharedMemBytes,
                                        strm,
                                        ofdmModKernelNodeParams.kernelParams,
                                        ofdmModKernelNodeParams.extra);
    assert(runStatus == hipSuccess);

    /**
     * @todo apply windowing effect, not used for now
     * 
     */
    // launch windowing kernel
    // if(m_pWindowCfg) // not nullptr
    // {
    //     const hipKernelNodeParams& windowKernelNodeParams = m_pWindowCfg->kernelNodeParamsDriver;
    //     hipError_t runStatus = hipModuleLaunchKernel(windowKernelNodeParams.func,
    //                                         windowKernelNodeParams.gridDimX,
    //                                         windowKernelNodeParams.gridDimY, 
    //                                         windowKernelNodeParams.gridDimZ,
    //                                         windowKernelNodeParams.blockDimX, 
    //                                         windowKernelNodeParams.blockDimY, 
    //                                         windowKernelNodeParams.blockDimZ,
    //                                         windowKernelNodeParams.sharedMemBytes,
    //                                         strm,
    //                                         windowKernelNodeParams.kernelParams,
    //                                         windowKernelNodeParams.extra);
    //     assert(runStatus == hipSuccess);
    // }
}

template <typename Tscalar, typename Tcomplex> 
void ofdmModulate<Tscalar, Tcomplex>::printTimeSample(int printLen)
{
    Tcomplex * temp_CPU_buffer = new Tcomplex[printLen];

    hipMemcpy(temp_CPU_buffer, m_ofdmModdynDescprCpu -> timeDataOut, printLen * sizeof(Tcomplex), hipMemcpyDeviceToHost);

    for (int index=0; index< printLen; index++)
    {
        printf("index: %d: %1.4e + %1.4e  i\n", index, float(temp_CPU_buffer[index].x), float(temp_CPU_buffer[index].y));
    }
    printf("Done printing output time domain signal from GPU \n");

    delete[] temp_CPU_buffer;
}