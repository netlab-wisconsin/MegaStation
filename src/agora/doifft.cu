#include "hip/hip_runtime.h"
/**
 * @file doifft.cc
 * @brief Implementation file for the DoIFFT class.
 */
#include "doifft.h"

#include "comms-lib.h"
#include "concurrent_queue_wrapper.h"
#include "datatype_conversion.h"
#include "logger.h"

static constexpr bool kPrintIFFTOutput = false;
static constexpr bool kPrintSocketOutput = false;
static constexpr bool kUseOutOfPlaceIFFT = false;
static constexpr bool kMemcpyBeforeIFFT = true;
static constexpr bool kPrintIfftStats = false;

DoIFFT::DoIFFT(Config* in_config, int in_tid,
               Table<complex_float>& in_dl_ifft_buffer,
               char* in_dl_socket_buffer,
               Table<hipStream_t>& cuda_streams,
               float2 *cuda_ifft_buffer,
               short *cuda_fft_out_buffer,
               Stats* in_stats_manager)
    : Doer(in_config, in_tid),
      dl_ifft_buffer_(in_dl_ifft_buffer),
      dl_socket_buffer_(in_dl_socket_buffer),
      fft_in_(cuda_ifft_buffer),
      fft_out_(cuda_fft_out_buffer),
      cuda_streams_(cuda_streams) {
  duration_stat_ = in_stats_manager->GetDurationStat(DoerType::kIFFT, in_tid);
  DftiCreateDescriptor(&mkl_handle_, DFTI_SINGLE, DFTI_COMPLEX, 1,
                       cfg_->OfdmCaNum());
  if (kUseOutOfPlaceIFFT) {
    DftiSetValue(mkl_handle_, DFTI_PLACEMENT, DFTI_NOT_INPLACE);
  }
  DftiCommitDescriptor(mkl_handle_);

  // Aligned for SIMD
  ifft_out_ = static_cast<float*>(
      Agora_memory::PaddedAlignedAlloc(Agora_memory::Alignment_t::kAlign64,
                                       2 * cfg_->OfdmCaNum() * sizeof(float)));
  ifft_shift_tmp_ = static_cast<complex_float*>(
      Agora_memory::PaddedAlignedAlloc(Agora_memory::Alignment_t::kAlign64,
                                       2 * cfg_->OfdmCaNum() * sizeof(float)));
  ifft_scale_factor_ = cfg_->OfdmCaNum();

  // GPU
  hipfftCreate(&cufft_plan_);
  hipfftPlan1d(&cufft_plan_, cfg_->OfdmCaNum(), HIPFFT_C2C, cfg_->BsAntNum());

  hipMemcpyFromSymbol(&hostLoadCallbackPtr,
      HIP_SYMBOL(cufftLoadCallbackIPtr),
      sizeof(hostLoadCallbackPtr));
  hipMemcpyFromSymbol(&hostStoreCallbackPtr,
      HIP_SYMBOL(cufftStoreCallbackIPtr),
      sizeof(hostStoreCallbackPtr));
  struct bothInfo cpu_info = {
    .ofdmStart = cfg_->OfdmDataStart(),
    .ofdmNum = cfg_->OfdmDataNum(),
    .ofdmCAnum = cfg_->OfdmCaNum(),
    .bsAnt = cfg_->BsAntNum(),
  };
  hipMalloc(reinterpret_cast<void **>(&stInfoPtr_), sizeof(struct bothInfo));
  hipMemcpy(stInfoPtr_, &cpu_info, sizeof(struct bothInfo),
    hipMemcpyHostToDevice);
  // hipfftXtSetCallback(cufft_plan_,
  //   reinterpret_cast<void **>(&hostLoadCallbackPtr),
  //   HIPFFT_CB_LD_COMPLEX, reinterpret_cast<void **>(&stInfoPtr_));
  hipfftXtSetCallback(cufft_plan_,
    reinterpret_cast<void **>(&hostStoreCallbackPtr),
    HIPFFT_CB_ST_COMPLEX, reinterpret_cast<void **>(&stInfoPtr_));
  fft_out_cpu_ = (short *)malloc(cfg_->OfdmCaNum() * cfg_->BsAntNum() * 2 * sizeof(short));
}

DoIFFT::~DoIFFT() {
  DftiFreeDescriptor(&mkl_handle_);
  std::free(ifft_out_);
  std::free(ifft_shift_tmp_);
}

EventData DoIFFT::Launch(size_t tag) {
  size_t start_tsc = GetTime::WorkerRdtsc();

  const size_t frame_id = gen_tag_t(tag).frame_id_;
  const size_t symbol_id = gen_tag_t(tag).symbol_id_;
  // const size_t ant_id = gen_tag_t(tag).ant_id_;

  const size_t symbol_idx_dl = cfg_->Frame().GetDLSymbolIdx(symbol_id);

  hipfftComplex *in_ptr = fft_in_ + symbol_idx_dl * cfg_->OfdmCaNum() * cfg_->BsAntNum();
  short *out_ptr = fft_out_ + 2 * symbol_idx_dl * cfg_->OfdmCaNum() * cfg_->BsAntNum();

  hipStream_t cur_stream = cuda_streams_[symbol_id][0];
  hipfftSetStream(cufft_plan_, cur_stream);

  const size_t start_tsc1 = GetTime::WorkerRdtsc();
  duration_stat_->task_duration_[1u] += start_tsc1 - start_tsc;

  hipfftExecC2C(cufft_plan_, in_ptr,
    reinterpret_cast<hipfftComplex *>(out_ptr), HIPFFT_BACKWARD);

  const size_t start_tsc2 = GetTime::WorkerRdtsc();
  duration_stat_->task_duration_[2u] += start_tsc2 - start_tsc1;

  // if (kDebugPrintInTask) {
  //   std::printf("In doIFFT thread %d: frame: %zu, symbol: %zu, antenna: %zu\n",
  //               tid_, frame_id, symbol_id, ant_id);
  // }

  // const size_t offset =
  //     (cfg_->GetTotalDataSymbolIdxDl(frame_id, symbol_idx_dl) *
  //      cfg_->BsAntNum()) +
  //     ant_id;

  // auto* ifft_in_ptr = reinterpret_cast<float*>(dl_ifft_buffer_[offset]);
  // auto* ifft_out_ptr =
  //     (kUseOutOfPlaceIFFT || kMemcpyBeforeIFFT) ? ifft_out_ : ifft_in_ptr;

  // std::memset(ifft_in_ptr, 0, sizeof(float) * cfg_->OfdmDataStart() * 2);
  // std::memset(ifft_in_ptr + (cfg_->OfdmDataStop()) * 2, 0,
  //             sizeof(float) * cfg_->OfdmDataStart() * 2);
  // CommsLib::FFTShift(reinterpret_cast<complex_float*>(ifft_in_ptr),
  //                    ifft_shift_tmp_, cfg_->OfdmCaNum());
  // if (kMemcpyBeforeIFFT) {
  //   std::memcpy(ifft_out_ptr, ifft_in_ptr,
  //               sizeof(float) * cfg_->OfdmCaNum() * 2);
  //   DftiComputeBackward(mkl_handle_, ifft_out_ptr);
  // } else {
  //   if (kUseOutOfPlaceIFFT) {
  //     // Use out-of-place IFFT here is faster than in place IFFT
  //     // There is no need to reset non-data subcarriers in ifft input
  //     // to 0 since their values are not changed after IFFT
  //     DftiComputeBackward(mkl_handle_, ifft_in_ptr, ifft_out_ptr);
  //   } else {
  //     DftiComputeBackward(mkl_handle_, ifft_in_ptr);
  //   }
  // }

  // bool clipping = false;
  // float max_abs = 0;
  // for (size_t i = 0; i < 2 * cfg_->OfdmCaNum(); i++) {
  //   float sample_val = ifft_out_ptr[i] / ifft_scale_factor_;
  //   if (sample_val >= 1) {
  //     clipping = true;
  //     break;
  //   }
  //   if (std::abs(sample_val) > max_abs) {
  //     max_abs = std::abs(sample_val);
  //   }
  // }
  // if (clipping) {
  //   AGORA_LOG_WARN("Clipping occured in Frame %zu, Symbol %zu, Antenna %zu\n",
  //                  frame_id, symbol_id, ant_id);
  // }
  // if (ant_id < cfg_->BfAntNum() && max_abs < 1e-4) {
  //   AGORA_LOG_WARN("Possibly bad antenna %zu with max sample value %2.2f\n",
  //                  ant_id, max_abs);
  // }
  // if (kPrintIfftStats) {
  //   std::printf("%2.3f\n", max_abs);
  // }

  // if (kPrintIFFTOutput) {
  //   std::stringstream ss;
  //   ss << "IFFT_output" << ant_id << "=[";
  //   for (size_t i = 0; i < cfg_->OfdmCaNum(); i++) {
  //     ss << std::fixed << std::setw(5) << std::setprecision(3)
  //        << dl_ifft_buffer_[offset][i].re << "+1j*"
  //        << dl_ifft_buffer_[offset][i].im << " ";
  //   }
  //   ss << "];" << std::endl;
  //   std::cout << ss.str();
  // }

  // auto* pkt = reinterpret_cast<Packet*>(
  //     &dl_socket_buffer_[offset * cfg_->DlPacketLength()]);
  // short* socket_ptr = &pkt->data_[2u * cfg_->OfdmTxZeroPrefix()];

  // // IFFT scaled results by OfdmCaNum(), we scale down IFFT results
  // // during data type coversion.  * 2 complex float -> float
  // SimdConvertFloatToShort(ifft_out_ptr, socket_ptr, cfg_->OfdmCaNum() * 2,
  //                         cfg_->CpLen() * 2, ifft_scale_factor_);

  hipMemcpyAsync(fft_out_cpu_, out_ptr,
    sizeof(short) * cfg_->OfdmCaNum() * cfg_->BsAntNum() * 2,
    hipMemcpyDeviceToHost, cur_stream);
  // hipComplex *fft_in_cpu_ = (hipComplex *)malloc(sizeof(hipComplex) * cfg_->OfdmCaNum() * cfg_->BsAntNum());
  // hipMemcpyAsync(fft_in_cpu_, in_ptr,
  //   sizeof(hipComplex) * cfg_->OfdmCaNum() * cfg_->BsAntNum(),
  //   hipMemcpyDeviceToHost, cur_stream);
  // hipStreamSynchronize(cur_stream);
  // if (symbol_idx_dl == 0) {// && (abs(fft_out_cpu_[0]) == 0 || abs(fft_out_cpu_[1]) == 0 || abs(fft_out_cpu_[2048]) == 0 || abs(fft_out_cpu_[2049]) == 0)) {
  //   spdlog::warn("[IFFT] ({},{}), ({},{}) -> ({},{}), ({},{})\n",
  //     fft_in_cpu_[0].x, fft_in_cpu_[0].y, fft_in_cpu_[1024].x, fft_in_cpu_[1024].y,
  //     fft_out_cpu_[cfg_->OfdmCaNum()*2+2], fft_out_cpu_[cfg_->OfdmCaNum()*2+3],
  //     fft_out_cpu_[cfg_->OfdmCaNum()*2+2050], fft_out_cpu_[cfg_->OfdmCaNum()*2+2051]);
  //   // for (size_t i = 0; i < cfg_->OfdmCaNum() * cfg_->BsAntNum(); i++) {
  //   //   spdlog::warn("IFFT input {}: ({},{}), output: ({},{})\n", i,
  //   //     in_ptr_cpu[i].x, in_ptr_cpu[i].y,
  //   //     fft_out_cpu_[i * 2], fft_out_cpu_[i * 2 + 1]);
  //   // }
  // }
  // free(fft_in_cpu_);
  for (size_t ant_id = 0; ant_id < cfg_->BsAntNum(); ant_id++) {
    const size_t offset =
      (cfg_->GetTotalDataSymbolIdxDl(frame_id, symbol_idx_dl) *
       cfg_->BsAntNum()) +
      ant_id;
    auto* pkt = reinterpret_cast<Packet*>(
      &dl_socket_buffer_[offset * cfg_->DlPacketLength()]);
    short* socket_ptr = &pkt->data_[2u * cfg_->OfdmTxZeroPrefix()];
    memcpy(socket_ptr + 2 * cfg_->CpLen(), fft_out_cpu_ + ant_id * cfg_->OfdmCaNum() * 2,
      sizeof(short) * cfg_->OfdmCaNum() * 2);
  }
  duration_stat_->task_duration_[3u] += GetTime::WorkerRdtsc() - start_tsc2;

  // if (kPrintSocketOutput) {
  //   std::stringstream ss;
  //   ss << "socket_tx_data" << ant_id << "_" << symbol_idx_dl << "=[";
  //   for (size_t i = 0; i < cfg_->SampsPerSymbol(); i++) {
  //     ss << socket_ptr[i * 2] << "+1j*" << socket_ptr[i * 2 + 1] << " ";
  //   }
  //   ss << "];" << std::endl;
  //   std::cout << ss.str();
  // }

  duration_stat_->task_count_++;
  duration_stat_->task_duration_[0u] += GetTime::WorkerRdtsc() - start_tsc;
  return EventData(EventType::kIFFT, tag);
}
