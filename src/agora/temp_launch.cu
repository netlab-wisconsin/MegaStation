#include "hip/hip_runtime.h"
#include "batched_gemv.h"
#include "set_kernel.h"

void demul_launch(
  unsigned int problem_size_row,
  unsigned int problem_size_col,
  unsigned int batch_count,
  unsigned int mod_func,
  const void *AMat,
  const void *BVec,
  signed char *CMat,
  unsigned long c_stride,
  unsigned long a_skip = 1,
  hipStream_t stream = nullptr) {
  BatchedGemv::Params params(
    {problem_size_row, problem_size_col},
    batch_count,
    mod_func,
    (myComplex *)AMat,
    (myComplex *)BVec,
    CMat,
    c_stride,
    a_skip
  );
  batched_gemv(params, stream);
}

template <typename T>
void set_ptr_launch(
  T **ptr_array,
  T *val,
  int num_ptrs,
  int inc,
  int skip,
  hipStream_t stream = nullptr) {
  dim3 block = dim3(32, 1, 1);
  dim3 grid = dim3((num_ptrs + block.x - 1) / block.x, 1, 1);

  set_pointer<T><<<grid, block, 0, stream>>>(ptr_array, val, num_ptrs, inc, skip);
}

template void set_ptr_launch<float2>(
  float2 **ptr_array,
  float2 *val,
  int num_ptrs,
  int inc,
  int skip,
  hipStream_t stream);